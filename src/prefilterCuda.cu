#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

#include <cutil_inline.h>
#include "prefilterCuda.h"

typedef struct {
	
	int numBlkSize;
	unsigned long int sizeSecQ;
	
	char * secuenciasQ_h;

	unsigned int * resultBlock_h;
	unsigned int * resultBlockId_h;
	
	char * secuenciasQ_d;

	unsigned int * resultBlock_d;
	unsigned int * resultBlockId_d;
	
	unsigned int * tamSecDb_d;
	char * secuenciasDB_d;
	
    //Stream for asynchronous command execution
    hipStream_t stream;
    
} GPUdata;

static const int gpuIndexes[5] = {0,1,2,3,4};

static const int PROTSIZE = 24;
static const int NUCLSIZE = 4;
__constant__ int PROTINDX[26] = {0/*A*/, 20/*B*/, 4/*C*/, 3/*D*/, 6/*E*/, 13/*F*/, 7/*G*/, 8/*H*/, 9/*I*/, 21/*J*/, 11/*K*/, 10/*L*/, 12/*M*/, 2/*N*/, 23/*O*/, 14/*P*/, 5/*Q*/, 1/*R*/, 15/*S*/, 16/*T*/, 23/*U*/, 19/*V*/, 17/*W*/, 23/*X*/, 18/*Y*/, 22/*Z*/};
__constant__ int NUCLINDX[26] = {0/*A*/, -1/*B*/, 1/*C*/, -1/*D*/, -1/*E*/, -1/*F*/, 2/*G*/, -1/*H*/, -1/*I*/, -1/*J*/, -1/*K*/, -1/*L*/, -1/*M*/, -1/*N*/, -1/*O*/, -1/*P*/, -1/*Q*/, -1/*R*/, -1/*S*/, 3/*T*/, -1/*U*/, -1/*V*/, -1/*W*/, -1/*X*/, -1/*Y*/, -1/*Z*/};

/*-------PROTOTYPES---------*/

__global__ void  cudaPrefilter(char * secQ, char * secDB, unsigned short int * tamanosDB, unsigned int * result, unsigned int * resultDB, int numDB, int tamanoQ, short int lmerLength);
void checkCUDAError(const char *msg);

/*--------------------------*/

//KERNEL 1
#define USE_SMEM_ATOMICS 0

#ifdef USE_SMEM_ATOMICS
	#ifdef CUDA_NO_SM13_ATOMIC_INTRINSICS
		#error Compilation target does not support shared-memory atomics
	#elif CUDA_NO_SM20_ATOMIC_INTRINSICS
		#error Compilation target does not support shared-memory atomics
	#else
		inline __device__ unsigned long long int suma(unsigned long long int * addr, unsigned long long int data)
		{
			return atomicAdd(addr, data);
		}
	#endif	
#else
	#error Shared-memory atomics not considered
#endif

#ifdef USE_SMEM_ATOMICS
	#ifdef CUDA_NO_SM13_ATOMIC_INTRINSICS
		#error Compilation target does not support shared-memory atomics
	#elif CUDA_NO_SM20_ATOMIC_INTRINSICS
		#error Compilation target does not support shared-memory atomics
	#else
		inline __device__ unsigned int suma2(unsigned int * addr, unsigned int data)
		{
			return atomicAdd(addr, data);
		}
	#endif	
#else
	#error Shared-memory atomics not considered
#endif

#ifdef USE_SMEM_ATOMICS
	#ifdef CUDA_NO_SM13_ATOMIC_INTRINSICS
		#error Compilation target does not support shared-memory atomics
	#elif CUDA_NO_SM20_ATOMIC_INTRINSICS
		#error Compilation target does not support shared-memory atomics
	#else
		inline __device__ unsigned int minimo(unsigned int * addr, unsigned int data)
		{
			return atomicMin(addr, data);
		}
	#endif	
#else
	#error Shared-memory atomics not considered
#endif

/**
* \fn void BlkToQuery (int numBloques, int numSecQ, unsigned int * tamSecQ, unsigned int *** qblock, unsigned short int ** indxQBlock, short int lmerLength)
* \brief Crea un array en el que contiene las relaciones entre las distintas querys y los bloques que las contienen.
* \param[in] numBloques, Numero de bloques totales.
* \param[in] numSecQ, Numero de secuencias totales.
* \param[in] tamSecQ, Array que contiene todos los tamaños de las secuencias.
* \param[in] qblock, Matriz que contiene los indices query que están contenidos en cada bloque.
* \param[in] indxQBlock, Array que contiene el número de secuencias query contenidas en casa bloque.
* \param[in] lmerLength, tamaño del lmer.
* \return .
*/
void BlkToQuery (int numBloques, int numSecQ, unsigned int * tamSecQ, unsigned int *** qblock, unsigned short int ** indxQBlock, short int lmerLength)
{
	int i, j, k;
	int contadorSec = 0;
	int contadorPos = 0;
	int contadorBlk = 0;
	int contadorAux = 0;
	int qStart = 0;

	(* qblock) = (unsigned int **) malloc (numBloques * sizeof(unsigned int *));
	(* indxQBlock) = (unsigned short int *) malloc (numBloques * sizeof(unsigned short int));
	
	for(i = 0; i < numSecQ; i++)
	{
		contadorSec++;
		for(j = 0; j < tamSecQ[i]; j++)
		{
			contadorPos++;
			
			if (contadorPos == TH_PER_BLOCK)
			{
				(* qblock)[contadorBlk] = (unsigned int *) malloc (contadorSec * sizeof(unsigned int));
				//Se inicializan el numero de secuencias de ese bloque.
				(* indxQBlock)[contadorBlk] = contadorSec;
				
				//Se meten las secuencias en la matriz.
				for(k = 0; k < contadorSec; k++)
				{
					contadorAux = qStart + k;
					(* qblock)[contadorBlk][k] = contadorAux;
				}
				
				contadorBlk++;
				contadorPos = 0;
				//Si se acaba ya la secuencia se usa la siguiente.
				if(j+1 ==  tamSecQ[i])
				{	
					contadorSec = 0;
					qStart = i+1;
				}
				else
				{
					contadorSec = 1;
					qStart = i;
				}
			}
		}
	}
	//Al terminar las secuencias se mete el último bloque.
	if(contadorPos > 0)
	{
		(* qblock)[contadorBlk] = (unsigned int *) malloc (contadorSec * sizeof(unsigned int));
		//Se inicializan el numero de secuencias de ese bloque.
		(* indxQBlock)[contadorBlk] = contadorSec;
		
		//Se meten las secuencias en la matriz.
		for(k = 0; k < contadorSec; k++)
		{
			contadorAux = qStart + k;
			(* qblock)[contadorBlk][k] = contadorAux;
		}
	}

	return;	
}

/**
* \fn __device__  unsigned int getHashIndx(char charParam, const int * indxParam)
* \brief Función que calcula el indice de un elemento del lmer.
* \param[in] charParam, Elemento del Lmer.
* \param[in] indxParam, Array de indices.
* \return indice del Lmer.
*/
__device__  unsigned int getHashIndx(char charParam, const int * indxParam) 
{
	unsigned int indx = 0;

	if (charParam >= 65 && charParam <= 90) {
		charParam -= 65;
		indx = indxParam[(int)charParam];
	} else if (charParam == 42) {
		indx = indxParam[23];
	} else {
		indx = indxParam[23];
	}

	return indx;
}

/**
* \fn __device__ int juan_hash (char * name, int length)
* \brief Función que calcula el valor hash de Nucleótidos o Proteínas.
* \param[in] name, Secuencia que contiene el Lmer.
* \param[in] length, Tamaño del lmer
* \return el valor hash del lmer.
*/
__device__ int juan_hash (char * name, int length) {
	short int i=0;
	int h = 0, h_aux = 0;

	for (i=0;i<length;i++) 
	{
		if ((name[i] == '*') || ((name[i] == 'U') && (length == 11)))
		{
			return -1;
		}
		else 
		{
			if (length == 3) 
			{
				h += getHashIndx(name[i], PROTINDX) * pow((double)PROTSIZE,length - (i + 1));
			}
			
			else if (length == 11) 
			{
				h_aux = getHashIndx(name[i], NUCLINDX) * pow((double)NUCLSIZE, length - (i + 1));
				if(h_aux != -1)
				{
					h += h_aux;
				}
				else
				{
					return -1;
				}
			}
		}
	}

	return h;
}

/**
* \fn unsigned char maskAnd(unsigned char value, int maskPos)
* \brief Función que saca de un byte el valor de un bit determinado por la posición.
* \param[in] value, Valor del que se debe sacar el dato.
* \param[in] maskPos, Máscara binaria para obtener el valor correcto.
* \return el valor del bit (1 o 0).
*/
__device__ unsigned int maskAnd(unsigned char value, int maskPos)
{
	unsigned char mask = 0;
	unsigned char result = 0;
	
	mask = pow((double) 2.0, maskPos);
	
	result = value & mask;
	
	if(result > 0)
		return 1;
	else
		return 0;
}

/**
* \fn unsigned char fromGuidToResult(unsigned char * bitValues, long int guid)
* \brief Función que devuelve un valor de la cadena de bits a partir de un GUID.
* \param[in] bitValues, Cadena de bytes comprimidos.
* \param[in] guid, Identificador Hash del valor buscado.
* \return valor buscado.
*/
__device__ unsigned int fromGuidToResult(unsigned char * bitValues, long int guid)
{
	int block = guid / BYTE_SIZE;
	int position = guid % BYTE_SIZE;
	
	return maskAnd(bitValues[block], position);
}

/**
* \fn __device__ unsigned int reduction(unsigned int * sdata, unsigned int tid)
* \brief Realiza una suma de datos usando el paralelismo de CUDA.
* \param[in] sdata, Array con todos los valores a sumar.
* \param[in] tid, Identificador de cada hilo.
* \return resultado de la suma.
*/
__device__ unsigned int reduction(unsigned int * sdata, unsigned int tid)
{	
	sdata[tid] += sdata[(tid + 128) % 256]; __syncthreads();
	sdata[tid] += sdata[(tid + 64) % 256]; __syncthreads();
	sdata[tid] += sdata[(tid + 32) % 256]; __syncthreads();
   
	sdata[tid] += sdata[(tid + 16) % 256];
	sdata[tid] += sdata[(tid +  8) % 256];
	sdata[tid] += sdata[(tid +  4) % 256];
	sdata[tid] += sdata[(tid +  2) % 256];
	sdata[tid] += sdata[(tid +  1) % 256];
	   
	if(tid == 0) 
	{
		return sdata[0];
	}
	
	return 0;
}

/**
* \fn __global__ void  cudaPrefilterProt (char * secQ, char * secDB, unsigned int * tamanosDB, unsigned int * result, unsigned int * resultDB, int numDB, unsigned long int tamanoQ, short int lmerLength, int scoreFilter)
* \brief Kernel Cuda para procesar Proteínas.
* \param[in] secQ, Secuencia con todas las secuencias query.
* \param[in] secDB, Secuencia con todas las secuencias de base de datos.
* \param[in] tamanosDB, Array que contiene todos los tamaños de las secuencias de base de datos.
* \param[in] result, Array que contiene el mejor Score obtenido por cada bloque.
* \param[in] resultDB, Array que contiene el identificador de la secuencia de base de datos con la que cada bloque ha obtenido el mejor Score.
* \param[in] numDB, Número de secuencias de base de datos.
* \param[in] tamanoQ, Tamaño total de todas las secuencias query.
* \param[in] lmerLength, tamaño del lmer.
* \param[in] scoreFilter, flag que discrimina que valor de score se debe devolver.
* \return .
*/
__global__ void  cudaPrefilterProt(char * secQ, char * secDB, unsigned int * tamanosDB, unsigned int * result, unsigned int * resultDB, int numDB, unsigned long int sizeDB, unsigned long int tamanoQ, short int lmerLength, int scoreFilter)
{
	unsigned int i;
	unsigned int scsMax = 0;
	double scsMax2 = 0;
	double pTotal = 0;	
	unsigned int dbIdMax = 0;
	unsigned int tid = threadIdx.x;
	unsigned int X = ((blockIdx.x * blockDim.x) + tid);
	long int guid = 0;
	
#if REDUCTION == 1
	__shared__ unsigned int sdata[256];
#else
	unsigned int scsAct;
#endif	

	__shared__ unsigned int scsHilo;
	__shared__ unsigned int scsHits;
	__shared__ char scsHitsFlag;
	__shared__ unsigned int valid_range;
	
	char letrita[3];
	
#if REDUCTION == 1
	sdata[tid] = 0;
#else
	scsAct = 0;
#endif

	i = 0;
	
	if (X < (tamanoQ - (lmerLength - 1)))
	{
		
		letrita[0] = secQ[X];
		letrita[1] = secQ[X + 1];
		letrita[2] = secQ[X + 2];
		
		valid_range = 257;
		
		if (letrita[2] == '*') {
			minimo(&valid_range, tid);
		}
		
		__syncthreads();
		
		guid = juan_hash(letrita, lmerLength);
		
		//Se recorren todas las db
		for (i = 0; i < numDB; i++)
		{
			scsHilo = 0;
			scsHits = 0;
			
			if (guid > -1) {
				if (guid >= sizeDB) {
#if REDUCTION == 0
					scsAct = 0;
#else
					sdata[tid] = 0;
#endif
				} else {
#if REDUCTION == 0
					scsAct = (unsigned int) secDB[(i*PROTDBSIZE) + guid];
#else
					sdata[tid] = (unsigned int) secDB[(i*PROTDBSIZE) + guid];
#endif
				}
			} else {
#if REDUCTION == 0
				scsAct = 0;
#else
				sdata[tid] = 0;
#endif
			}
			
			if (scsHitsFlag == 0) {
#if REDUCTION == 0
				if (scsAct > 1) {
					scsAct = 0;
				}
#else
				if (sdata[tid] > 1) {
					sdata[tid] = 0;
				}
#endif
			}
			
#if REDUCTION == 0
			// Se realiza la suma de los valores de cada hilo
			if (scsAct > 1)
			{
				// ATOMIC ADD
				scsHitsFlag = 1;
				suma2(&scsHilo, scsAct);
				if (scoreFilter != 2) 
				{
					suma2(&scsHits, 1);
				}
			} else { 
				//ATOMIC ADD
				suma2(&scsHilo, scsAct);
			}
#else
			if (sdata[tid] > 1) {
				scsHitsFlag = 1;
				// REDUCTION
				scsHilo = reduction(sdata, tid);
				if (scoreFilter != 2) {
					suma2(&scsHits, 1);
				}
			} else {
				// REDUCTION
				scsHilo = reduction(sdata, tid);
			}
#endif

			__syncthreads();			
			
			if (tid == 0) 
			{
				pTotal = scsHilo * 1000; 
				
				//if (tamanosDB[i] >= TH_PER_BLOCK)
				//{
				//	pTotal = pTotal / (tamanosDB[i] / (TH_PER_BLOCK * 1.0));
				//}
				
				// *** Elegir del mejor resultado
				if (pTotal >= scsMax2)
				{
					scsMax2 = pTotal;
					dbIdMax = i;
					
					// Caso Hits
					if(scsHitsFlag == 0)
					{
						scsMax = scsHilo;
					}
					else
					{
						// Caso Scores Promedio
						if (scoreFilter == 0)
						{
							scsMax = (scsHilo / scsHits);
						}
						// Caso Raw Scores con missmatch
						else if (scoreFilter == 1) 
						{
							// Mayor número de missmatch que de match
							if (((valid_range - 1) - scsHits) > scsHilo) 
							{
								scsMax = 0;
							} 
							// Mayor número de match que de missmatch
							else 
							{
								scsMax = scsHilo - (((valid_range - 1) - scsHits) * MISMATCH_PENALTY);
							}
						}
						// Caso Raw Scores 
						else if (scoreFilter == 2) 
						{
							scsMax = scsHilo;
						}
					}
				}
			}
		
			__syncthreads();
		}
		
		if(tid == 0)
		{			
			result[blockIdx.x] = scsMax;
			resultDB[blockIdx.x] = dbIdMax;
		}
	}

	return;
}

/**
* \fn __global__ void  cudaPrefilterNucl(char * secQ, char * secDB, unsigned int * tamanosDB, unsigned int * result, unsigned int * resultDB, int numDB, unsigned long int tamanoQ, short int lmerLength)
* \brief Kernel Cuda para procesar Nucleótidos
* \param[in] secQ, Secuencia con todas las secuencias query.
* \param[in] secDB, Secuencia con todas las secuencias de base de datos.
* \param[in] tamanosDB, Array que contiene todos los tamaños de las secuencias de base de datos.
* \param[in] result, Array que contiene el mejor Score obtenido por cada bloque.
* \param[in] resultDB, Array que contiene el identificador de la secuencia de base de datos con la que cada bloque ha obtenido el mejor Score.
* \param[in] numDB, Número de secuencias de base de datos.
* \param[in] tamanoQ, Tamaño total de todas las secuencias query.
* \param[in] lmerLength, tamaño del lmer.
* \return .
*/
__global__ void  cudaPrefilterNucl(char * secQ, char * secDB, unsigned int * tamanosDB, unsigned int * result, unsigned int * resultDB, int numDB, unsigned long int sizeDB, unsigned long int tamanoQ, short int lmerLength)
{
	unsigned int i;
	unsigned int scsAct = 0;
	//unsigned int sumaPosDB = 0;
	unsigned int scsMax = 0;
	double scsMax2 = 0;
	double pTotal = 0;
	unsigned int scsAux = 0;	
	unsigned int dbIdMax = 0;
	unsigned short int tid = threadIdx.x;
	unsigned int X = ((blockIdx.x * blockDim.x) + tid);
	long int guid = 0;
	
	__shared__ unsigned int scsHilo;
	
	char letritaNucl[11];
				
	if (X < (tamanoQ - (lmerLength - 1))) {		
		letritaNucl[0] = secQ[X];
		letritaNucl[1] = secQ[X + 1];
		letritaNucl[2] = secQ[X + 2];
		letritaNucl[3] = secQ[X + 3];
		letritaNucl[4] = secQ[X + 4];
		letritaNucl[5] = secQ[X + 5];
		letritaNucl[6] = secQ[X + 6];
		letritaNucl[7] = secQ[X + 7];
		letritaNucl[8] = secQ[X + 8];
		letritaNucl[9] = secQ[X + 9];
		letritaNucl[10] = secQ[X + 10];
		
		//Se recorren todas las db
		for (i = 0; i < numDB; i++) {
			scsAct = 0;
			scsHilo = 0;
					
			guid = juan_hash (letritaNucl, lmerLength);
			if (guid != -1)
			{
				scsAct = fromGuidToResult((unsigned char *) &secDB[i*NUCL_BIT_SIZE], guid);
			}
		 
			//Se realiza la suma de los valores de cada hilo mediante atomicAdd.
			suma2(&scsHilo, scsAct);
			
			__syncthreads();
			
			if(tid == 0)
			{
				scsAux = scsHilo * 1000;

				if(scsHilo == TH_PER_BLOCK)
				{
					pTotal = scsAux;
				}
				else if(tamanosDB[i] >= TH_PER_BLOCK)
				{
					pTotal = scsAux / (tamanosDB[i] / (TH_PER_BLOCK * 1.0));
				}
				else
				{
					pTotal = scsAux;
				}
				
				if(pTotal >= scsMax2)
				{
					scsMax2 = pTotal;
					scsMax = scsHilo;
					dbIdMax = i;
				}
			}			
			__syncthreads();
		}
		
		if(tid == 0)
		{	
			result[blockIdx.x] = scsMax;
			resultDB[blockIdx.x] = dbIdMax;
		}
	}

	return;
}

/**
* \fn extern "C" void prefilterCuda(char * secuenciasQ, char * secuenciasDB, int numSecDB, int numSecQ, unsigned int * tamSecDb, unsigned int * tamSecQ, unsigned long int sizeSecDB, unsigned long int sizeSecQ, short int lmerLength, char flagInicializacion, char * out, unsigned int *** qblock, unsigned short int ** indxQBlock, unsigned int * numBloques, unsigned int ** resultBlock, unsigned int ** resultBlockId, int gpuCard, int scoreFilter)
* \brief Función que prepara, reserva e inicializa las estructuras y arrays necesarias para la ejecución de CUDA.
* \param[in] secuenciasQ, Secuencia con todas las secuencias query.
* \param[in] secuenciasDB, Secuencia con todas las secuencias de base de datos.
* \param[in] numSecDB, Número de secuencias de base de datos.
* \param[in] numSecQ, Número de secuencias query.
* \param[in] tamSecDb, Array que contiene todos los tamaños de las secuencias de base de datos.
* \param[in] tamSecQ, Array que contiene todos los tamaños de las secuencias de base de datos.
* \param[in] sizeSecDB, Tamaño total de todas las secuencias de la base de datos.
* \param[in] sizeSecQ, Tamaño total de todas las secuencias query.
* \param[in] lmerLength, Tamaño del Lmer.
* \param[in] flagInicializacion, Flag que indica si el sistema ya se ha inicializado (No se usa).
* \param[in] out, Nombre del fichero de salida (No se usa).
* \param[in] qblock, Matriz que contiene los indices query que están contenidos en cada bloque.
* \param[in] indxQBlock, Array que contiene el número de secuencias query contenidas en casa bloque.
* \param[in] numBloques, Número de bloques que se crearán en la ejecución del Kernel CUDA.
* \param[in] resultBlock, Array que contiene el mejor Score obtenido por cada bloque.
* \param[in] resultBlockId, Array que contiene el identificador de la secuencia de base de datos con la que cada bloque ha obtenido el mejor Score.
* \param[in] gpuCard, Número de Gpus a usar (No se usa).
* \param[in] scoreFilter, flag que discrimina que valor de score se debe devolver.
* \return .
*/
extern "C" void prefilterCuda(char * secuenciasQ, char * secuenciasDB, int numSecDB, int numSecQ, unsigned int * tamSecDb, unsigned int * tamSecQ, unsigned long int sizeSecDB, unsigned long int sizeSecQ, 
								short int lmerLength, char flagInicializacion, char * out, unsigned int *** qblock, unsigned short int ** indxQBlock, unsigned int * numBloques,
								unsigned int ** resultBlock, unsigned int ** resultBlockId, int gpuCard, int scoreFilter)
{
	struct timeval iniCuda, finCuda;
	struct timeval ini1, fin1;
	struct timeval ini2, fin2;
	struct timeval ini3, fin3;
	
	int device_count = 0;
	int device_index = 0;
	
	GPUdata paralelData[5];
	int acumulador = 0;
	int acumulador2 = 0;
			
	//Inicialización de la GPU
	hipGetDeviceCount(&device_count);
	device_count = 4;
	
	if (numSecQ == 1)
		device_count = 1;

	fprintf(stderr,"Device Numbers: %d\n", device_count);
	
	//Cálculo del numero de bloques que se van a ejecutar.
	if ((sizeSecQ % TH_PER_BLOCK) == 0)
		(*numBloques) = sizeSecQ / TH_PER_BLOCK;
	else
		(*numBloques) = (sizeSecQ / TH_PER_BLOCK) + 1;

	fprintf(stderr,"numBloques %d sizeQ %ld sizeDB %ld\n", (*numBloques), sizeSecQ, sizeSecDB);	
	
	(*resultBlock) = (unsigned int *) malloc ((*numBloques) * sizeof(unsigned int));
	(*resultBlockId) = (unsigned int *) malloc ((*numBloques) * sizeof(unsigned int));
	
	acumulador = 0;
	
	//Inicialización del numero de bloques que va a ejecutar cada GPU
	//Inicialización de las posiciones de memoria de los arrays de entrada para cada GPU.
	//Creación de el hilo de envío de cada GPU.
	for (device_index = 0; device_index < device_count; device_index++)
	{
		hipSetDevice(gpuIndexes[device_index]);
		
		paralelData[device_index].numBlkSize = (*numBloques) / device_count;
		
		if(acumulador + (((*numBloques) / device_count) * TH_PER_BLOCK) > sizeSecQ)
		{
			paralelData[device_index].sizeSecQ = sizeSecQ - acumulador;
		} else {
			paralelData[device_index].sizeSecQ = ((*numBloques) / device_count) * TH_PER_BLOCK;
		}
		
		paralelData[device_index].secuenciasQ_h = secuenciasQ + acumulador;

		paralelData[device_index].resultBlock_h = (*resultBlock) + acumulador2;
		paralelData[device_index].resultBlockId_h = (*resultBlockId) + acumulador2;

#if DEBUG == 1
		fprintf(stderr,"leido desde bloque %d hasta %d, leido desde sizeSecQ %d hasta %d GPU : %d\n", 
				acumulador2, ((*numBloques) / device_count) + acumulador2, acumulador, 
				paralelData[device_index].sizeSecQ + acumulador, device_index);
#endif
		
		acumulador += ((*numBloques) / device_count) * TH_PER_BLOCK;
		acumulador2 += (*numBloques) / device_count;
		
		hipStreamCreate(&paralelData[device_index].stream);
	}
	
	//El último bloque lo procesa la última GPU (en caso de no ser una división esacta).
	int restoBloques = ((*numBloques) % device_count);
	if (restoBloques != 0)
	{
		paralelData[device_index-1].numBlkSize += restoBloques;
		paralelData[device_index-1].sizeSecQ += sizeSecQ - acumulador;

#if DEBUG == 1		
		fprintf(stderr, "anadidos %d bloques adicionales, GPU: %d desde: %d hasta: %d\n", restoBloques, device_index-1, acumulador, acumulador + (sizeSecQ - acumulador));
#endif
	}
				
	gettimeofday(&iniCuda, NULL);
	
	//Generar relación bloques query
	BlkToQuery ((*numBloques), numSecQ, tamSecQ, qblock, indxQBlock, lmerLength);
	
	if (numSecDB <= NUM_SEC_DB_PROCESS)
	{
		gettimeofday(&ini1, NULL);
		
		//Reserva de memoria de los datos GPU.	
		for (device_index = 0; device_index < device_count; device_index++)
		{
			hipSetDevice(gpuIndexes[device_index]);
			
			// *** Reserva memoria CUDA -- BASE DE DATOS.
			hipMalloc((void**) &paralelData[device_index].secuenciasDB_d, sizeSecDB * sizeof(char));
			hipMalloc((void**) &paralelData[device_index].tamSecDb_d, numSecDB * sizeof(unsigned int));
			checkCUDAError("cuda malloc db");
			
			// *** Reserva de memoria CUDA -- QUERIES Y RESULTADOS
			hipMalloc((void**) &paralelData[device_index].secuenciasQ_d, paralelData[device_index].sizeSecQ * sizeof(char));
			hipMalloc((void**) &paralelData[device_index].resultBlock_d, paralelData[device_index].numBlkSize * sizeof(unsigned int));
			hipMalloc((void**) &paralelData[device_index].resultBlockId_d, paralelData[device_index].numBlkSize * sizeof(unsigned int));
			checkCUDAError("cuda malloc q");
			
			// *** Copia de datos a CUDA	
			// NOTA: Se ha usado el mismo array para copiar la DB en todas las GPU.
			hipMemcpyAsync(paralelData[device_index].secuenciasDB_d, secuenciasDB,  sizeSecDB * sizeof(char), hipMemcpyHostToDevice, paralelData[device_index].stream);
			hipMemcpyAsync(paralelData[device_index].tamSecDb_d, tamSecDb,  numSecDB * sizeof(unsigned int), hipMemcpyHostToDevice, paralelData[device_index].stream);
			
				// SYNC MODE
				//hipMemcpy(paralelData[device_index].secuenciasDB_d, secuenciasDB, sizeSecDB * sizeof(char), hipMemcpyHostToDevice);
				//hipMemcpy(paralelData[device_index].tamSecDb_d, tamSecDb, numSecDB * sizeof(unsigned int), hipMemcpyHostToDevice);
			
			checkCUDAError("cuda memcpy Send BD");
			
			hipMemcpyAsync(paralelData[device_index].secuenciasQ_d, paralelData[device_index].secuenciasQ_h, paralelData[device_index].sizeSecQ * sizeof(char), hipMemcpyHostToDevice, paralelData[device_index].stream);
			 		 
				// SYNC MODE
				//hipMemcpy(paralelData[device_index].secuenciasQ_d, paralelData[device_index].secuenciasQ_h, paralelData[device_index].sizeSecQ * sizeof(char), hipMemcpyHostToDevice);
			
			checkCUDAError("cuda memcpy Send Q");
		}
			
		gettimeofday(&fin1, NULL);
#if DEBUG == 1
		fprintf(stderr,"Time CUDA send: %ld msec ---- ---- \n", (((fin1.tv_sec*1000000)+fin1.tv_usec)-((ini1.tv_sec*1000000)+ini1.tv_usec))/1000);
#endif
		
		gettimeofday(&ini2, NULL);
			
		//Inicialización de los hilos y bloques para CUDA, y lanzamiento del Kernel	
		for (device_index = 0; device_index < device_count; device_index++)
		{
			hipSetDevice(gpuIndexes[device_index]);
			
			dim3 dimBlock(TH_PER_BLOCK, 1);
			dim3 dimGrid(paralelData[device_index].numBlkSize, 1);

			
			//Kernel
			if(lmerLength == 3)
			{
				cudaPrefilterProt <<< dimGrid, dimBlock, 0, paralelData[device_index].stream >>>
				(paralelData[device_index].secuenciasQ_d, paralelData[device_index].secuenciasDB_d, 
				paralelData[device_index].tamSecDb_d, paralelData[device_index].resultBlock_d, 
				paralelData[device_index].resultBlockId_d, numSecDB, sizeSecDB, paralelData[device_index].sizeSecQ, lmerLength, scoreFilter);
			}
			else if (lmerLength == 11)
			{
				cudaPrefilterNucl <<< dimGrid, dimBlock, 0, paralelData[device_index].stream >>>
				(paralelData[device_index].secuenciasQ_d, paralelData[device_index].secuenciasDB_d, 
				paralelData[device_index].tamSecDb_d, paralelData[device_index].resultBlock_d, 
				paralelData[device_index].resultBlockId_d, numSecDB, sizeSecDB, paralelData[device_index].sizeSecQ, lmerLength);
			}
		}
		
		for (device_index = 0; device_index < device_count; device_index++)
		{
			// *** Puede quitarse en modo release (únicamente afecta en la medida de tiempos no en el rendimiento)
			hipDeviceSynchronize();																		   
			checkCUDAError("kernel invocation");
		}
		
		gettimeofday(&fin2, NULL);
#if DEBUG == 1
		fprintf(stderr,"Time CUDA kernel: %ld msec ---- ---- \n", (((fin2.tv_sec*1000000)+fin2.tv_usec)-((ini2.tv_sec*1000000)+ini2.tv_usec))/1000);
#endif
			
		gettimeofday(&ini3, NULL);
		
		//Recuperación de los resultados.
		for (device_index = 0; device_index < device_count; device_index++)	
		{	
			hipSetDevice(gpuIndexes[device_index]);
			
			hipMemcpyAsync(paralelData[device_index].resultBlock_h, paralelData[device_index].resultBlock_d, paralelData[device_index].numBlkSize * sizeof(unsigned int), hipMemcpyDeviceToHost, paralelData[device_index].stream);
			hipMemcpyAsync(paralelData[device_index].resultBlockId_h, paralelData[device_index].resultBlockId_d, paralelData[device_index].numBlkSize * sizeof(unsigned int), hipMemcpyDeviceToHost, paralelData[device_index].stream);
			
			//hipMemcpy(paralelData[device_index].resultBlock_h, paralelData[device_index].resultBlock_d, paralelData[device_index].numBlkSize * sizeof(unsigned int), hipMemcpyDeviceToHost);
			//hipMemcpy(paralelData[device_index].resultBlockId_h, paralelData[device_index].resultBlockId_d, paralelData[device_index].numBlkSize * sizeof(unsigned int), hipMemcpyDeviceToHost);
			
			checkCUDAError("cuda memcpy Recv");
		}

#if DEBUG == 1
		fprintf(stderr,"Synchronizing Streams\n");
#endif
		acumulador = 0;
		//Se juntan todas los resultados parciales.
		for (device_index = 0; device_index < device_count; device_index++)
		{
			hipSetDevice(gpuIndexes[device_index]);
			checkCUDAError("hipSetDevice");
			
			hipStreamSynchronize(paralelData[device_index].stream);
			checkCUDAError("hipStreamSynchronize");
			
			// *** Remove to avoid overlapping
			// memcpy((*resultBlock) + acumulador, paralelData[device_index].resultBlock_h, paralelData[device_index].numBlkSize * sizeof(unsigned int));
			// memcpy((*resultBlockId) + acumulador, paralelData[device_index].resultBlockId_h, paralelData[device_index].numBlkSize * sizeof(unsigned int));
			acumulador += paralelData[device_index].numBlkSize;
		}
		
		gettimeofday(&fin3, NULL);
#if DEBUG == 1		
		fprintf(stderr,"Time CUDA recv: %ld msec ---- ---- \n", (((fin3.tv_sec*1000000)+fin3.tv_usec)-((ini3.tv_sec*1000000)+ini3.tv_usec))/1000);
#endif
	}
	// *** El numero de secuencias BD es demasiado grande
	else
	{
		fprintf(stderr,"Demasiadas secuencias de Base de Datos\n");
		exit(0);
	}
		
	//Liberación memoria CUDA
	for (device_index = 0; device_index < device_count; device_index++)
	{
#if DEBUG == 1
		fprintf(stderr,"Liberando recursos del device %d\n",device_index);
#endif
		hipSetDevice(gpuIndexes[device_index]);
		checkCUDAError("hipSetDevice");
		
		if (paralelData[device_index].secuenciasQ_d)
			hipFree(paralelData[device_index].secuenciasQ_d);
		if (paralelData[device_index].resultBlock_d)
			hipFree(paralelData[device_index].resultBlock_d);
		if (paralelData[device_index].resultBlockId_d)
			hipFree(paralelData[device_index].resultBlockId_d);
		
		if (paralelData[device_index].tamSecDb_d)
			hipFree(paralelData[device_index].tamSecDb_d);
		if (paralelData[device_index].secuenciasDB_d)
			hipFree(paralelData[device_index].secuenciasDB_d);
			
		checkCUDAError("free memory");
		hipStreamDestroy(paralelData[device_index].stream);
		
		hipDeviceReset();
	}
	
	gettimeofday(&finCuda, NULL);
	
#if DEBUG == 1
	fprintf(stderr,"Time CUDA: %ld msec ---- ---- \n", (((finCuda.tv_sec*1000000)+finCuda.tv_usec)-((iniCuda.tv_sec*1000000)+iniCuda.tv_usec))/1000);
#endif
	
	return;
}

/**
* \fn void sys_call(char * sysdata)
* \brief Función que realiza una llamada al sistema con los datos pasados.
* \param[in] sysdata, Datos para la llamada al sistema.
* \return .
*/
void report_error(char * sysdata)
{
	char systemCall[200];
	int size = 0;
	
	if (sysdata)
	{
		memset(systemCall, 0, 200 * sizeof(char));
		size = strlen(sysdata);
		memcpy(systemCall, sysdata, size*sizeof(char));
		system((const char *) systemCall);
	}
}

/**
* \fn void checkCUDAError(const char *msg)
* \brief Función que evalua si ha habido algún error de ejecucion en CUDA.
* \param[in] msg, Mensaje devuelto por la librería de CUDA.
* \return .
*/
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    int flagError = 0, device_index = 0, device_count = 4;
    char aux[100];
    
    if (hipErrorPriorLaunchFailure == err)
    {
    	memset(aux, 0, 100 * sizeof(char));
#if LOCAL_TEST == 0
		sprintf(aux , "blast_reporter \"Cuda error: hipErrorPriorLaunchFailure: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#else
		sprintf(aux , "./blast_reporter \"Cuda error: hipErrorPriorLaunchFailure: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#endif
		report_error(aux);
		fprintf(stderr, "Cuda error: hipErrorPriorLaunchFailure: %s: %s.\n", msg, hipGetErrorString(err) );
        flagError = 1;
	}
	if (hipErrorLaunchTimeOut == err)
    {
    	memset(aux, 0, 100 * sizeof(char));
#if LOCAL_TEST == 0
		sprintf(aux , "blast_reporter \"Cuda error: hipErrorLaunchTimeOut: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#else
		sprintf(aux , "./blast_reporter \"Cuda error: hipErrorLaunchTimeOut: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#endif
		report_error(aux);
		fprintf(stderr, "Cuda error: hipErrorLaunchTimeOut: %s: %s.\n", msg, hipGetErrorString(err) );
        flagError = 1;
	}
	if (hipErrorInvalidDeviceFunction == err)
    {
    	memset(aux, 0, 100 * sizeof(char));
#if LOCAL_TEST == 0
		sprintf(aux ,"blast_reporter \"Cuda error: hipErrorInvalidDeviceFunction: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#else
		sprintf(aux ,"./blast_reporter \"Cuda error: hipErrorInvalidDeviceFunction: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#endif	
		report_error(aux);
		fprintf(stderr, "Cuda error: hipErrorInvalidDeviceFunction: %s: %s.\n", msg, hipGetErrorString(err) );
        flagError = 1;
	}
	if (hipErrorInvalidValue == err)
    {
    	memset(aux, 0, 100 * sizeof(char));
#if LOCAL_TEST == 0
		sprintf(aux ,"blast_reporter \"Cuda error: hipErrorInvalidValue: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#else
		sprintf(aux ,"./blast_reporter \"Cuda error: hipErrorInvalidValue: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#endif
		report_error(aux);
		fprintf(stderr, "Cuda error: hipErrorInvalidValue: %s: %s.\n", msg, hipGetErrorString(err) );
        flagError = 1;
	}
	
    if (hipSuccess != err) {
    	memset(aux, 0, 100 * sizeof(char));
#if LOCAL_TEST == 0
		sprintf(aux ,"blast_reporter \"Cuda error: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#else
		sprintf(aux ,"./blast_reporter \"Cuda error: %s: %s.\" %c", msg, hipGetErrorString(err),'\0');
#endif
		report_error(aux);
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
        flagError = 1;
    }
    
    if (flagError == 1) {
    	//Liberación memoria CUDA
		for (device_index = 0; device_index < device_count; device_index++)
		{
#if DEBUG == 1
			fprintf(stderr,"Liberando recursos del device %d\n",device_index);
#endif
			hipError_t ret = hipSetDevice(gpuIndexes[device_index]);
			if (ret != hipSuccess) {
				fprintf(stderr,"\t%s\n",hipGetErrorString(ret));
			}
			
			ret = hipDeviceReset();
			if (ret != hipSuccess) {
				fprintf(stderr,"\t%s\n",hipGetErrorString(ret));
			}
		}
		
		exit(-1);
    }
}
